
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void vector(float *A, float *B, float *C, int N){
    int i= blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N){
        C[i] = A[i] + B[i];
    }
}

int main() {
    int N = 4;
    size_t size = N * sizeof(float);
    float A[] = {1,2,3,4};
    float B[] = {5,6,7,8};
    float C[4];

    float *d_A, *d_B, *d_C;

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A,A,size,hipMemcpyHostToDevice);
    hipMemcpy(d_B,B,size,hipMemcpyHostToDevice);

    vector<<<1, N>>>(d_A, d_B, d_C, N);

    hipMemcpy(C,d_C, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++){
        printf("%f\t", C[i]);
    }
}