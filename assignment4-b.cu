// Assignment 4
// Write a CUDA Program for : 
// 2. Matrix Multiplication using CUDA C 


#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void matMul(float *A, float *B, float *C, int N){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x; 

    if (row < N && col < N){
        float sum = 0;
        for (int  k = 0; k < N;k++){
            sum = sum + A[row * N + k] * B[N * k + col];
        }
        C[row * N + col]  = sum;
    }
}

int main() {
    int N = 2;
    size_t size = N * N * sizeof(float);
    float A[] = {1,2,3,4};
    float B[] = {5,6,7,8};
    float C[4];

    float *d_A, *d_B, *d_C;

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A,A,size,hipMemcpyHostToDevice);
    hipMemcpy(d_B,B,size,hipMemcpyHostToDevice);

    dim3 blocks(N,N);
    dim3 threads(1,1);

    matMul<<<blocks, threads>>>(d_A, d_B, d_C, N);

    hipMemcpy(C,d_C, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < N * N; i++){
        printf("%f\t", C[i]);
    }
}